
#include <hip/hip_runtime.h>
/***********************************************************************
An attempt at putting the ray-casting operation into CUDA
An attempt at putting the ray-casting operation into CUDA

Author: Matthew Turk <matthewturk@gmail.com>
Affiliation: KIPAC/SLAC/Stanford
Homepage: http://yt.enzotools.org/
License:
  Copyright (C) 2009 Matthew Turk.  All Rights Reserved.

  This file is part of yt.

  yt is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
***********************************************************************/

extern __shared__ float array[];

#define NUM_SAMPLES 5
#define VINDEX(A,B,C) data[((((A)+ci[0])*(ds[1]+1)+((B)+ci[1]))*(ds[2]+1)+ci[2]+(C))]

#define fmin(A, B) ( (A < B) ? A : B )
#define fmax(A, B) ( (A > B) ? A : B )
#define fclip(A, B, C) ( fmax( fmin(A, C), B) )

struct transfer_function
{
    float *vs[4];
    float dbin;
    float bounds[2];
};

struct grid
{
    float left_edge[3];
    float right_edge[3];
    float dds[3];
    int dims[3];
    float *data;
};

__device__ float interpolate(float *data, int ds[3], int *ci, float *dp)
{
    int i;
    float dv, dm[3];
    for(i=0;i<3;i++)dm[i] = (1.0 - dp[i]);
    dv  = 0.0;
    dv += VINDEX(0,0,0) * (dm[0]*dm[1]*dm[2]);
    dv += VINDEX(0,0,1) * (dm[0]*dm[1]*dp[2]);
    dv += VINDEX(0,1,0) * (dm[0]*dp[1]*dm[2]);
    dv += VINDEX(0,1,1) * (dm[0]*dp[1]*dp[2]);
    dv += VINDEX(1,0,0) * (dp[0]*dm[1]*dm[2]);
    dv += VINDEX(1,0,1) * (dp[0]*dm[1]*dp[2]);
    dv += VINDEX(1,1,0) * (dp[0]*dp[1]*dm[2]);
    dv += VINDEX(1,1,1) * (dp[0]*dp[1]*dp[2]);
    return dv;
}

__device__ void eval_transfer(float dt, float dv, float rgba[4],
                               transfer_function tf)
{
    int i, bin_id;
    float temp, bv, dy, dd, ta;

    bin_id = (int) ((dv - tf.bounds[0]) / tf.dbin);
    bv = tf.vs[3][bin_id  ];
    dy = tf.vs[3][bin_id+1] - bv;
    dd = dv - (tf.bounds[0] + bin_id*tf.dbin);
    temp = bv+dd*(dy/tf.dbin);
    ta = temp;
    for (i = 0; i < 3; i++)
    {
        bv = tf.vs[i][bin_id  ];
        dy = tf.vs[i][bin_id+1];
        dd = dv - (tf.bounds[0] + bin_id*tf.dbin);
        temp = bv+dd*(dy/tf.dbin);
        rgba[i] += (1.0 - rgba[3])*ta*temp*dt;
    }
    rgba[3] += (1.0 - rgba[3])*ta*dt;
}

__device__ void sample_values(float v_pos[3], float v_dir[3],
                   float enter_t, float exit_t, int ci[3], float rgba[4],
                   transfer_function tf, grid tg)
{
    float cp[3], dp[3], dt, t, dv;
    int dti, i;
    dt = (exit_t - enter_t) / (NUM_SAMPLES-1);
    for (dti = 0; dti < NUM_SAMPLES - 1; dti++)
    {
        t = enter_t + dt*dti;
        for (i = 0; i < 3; i++)
        {
            cp[i] = v_pos[i] + t * v_dir[i];
            dp[i] = fclip(fmod(cp[i], tg.dds[i])/tg.dds[i], 0.0, 1.0);
        }
        dv = interpolate(tg.data, tg.dims, ci, dp);
        eval_transfer(dt, dv, rgba, tf);
    }
}
                   

/* We need to know several things if we want to ray cast through a grid.
   We need the grid spatial information, as well as its values.  We also need
   the transfer function, which defines what our image will look like. */

__global__ void ray_cast(float *grid_data,
                         int dims[3],
                         float left_edge[3],
                         float dds[3],
                         float tf_r[255],
                         float tf_g[255],
                         float tf_b[255],
                         float tf_a[255],
                         float tf_bounds[2],
                         float v_dir[3],
                         float *av_pos,
                         float *image_r,
                         float *image_g,
                         float *image_b,
                         float *image_a)
{

    transfer_function tf;
    tf.vs[0] = tf_r;
    tf.vs[1] = tf_g;
    tf.vs[2] = tf_b;
    tf.vs[3] = tf_a;
    tf.bounds[0] = tf_bounds[0]; tf.bounds[1] = tf_bounds[1];
    tf.dbin = (tf.bounds[1] - tf.bounds[0])/255.0;

    /* Set up the grid, just for convenience */
    grid tg;

    tg.dims[0] = dims[0];
    tg.dims[1] = dims[1];
    tg.dims[2] = dims[2];

    tg.left_edge[0] = left_edge[0];
    tg.left_edge[1] = left_edge[1];
    tg.left_edge[2] = left_edge[2];

    tg.right_edge[0] = left_edge[0] + dims[0] * dds[0];
    tg.right_edge[1] = left_edge[1] + dims[1] * dds[1];
    tg.right_edge[2] = left_edge[2] + dims[2] * dds[2];

    tg.dds[0] = dds[0];
    tg.dds[1] = dds[1];
    tg.dds[2] = dds[2];

    float rgba[4];
    
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    rgba[0] = image_r[idx1];
    rgba[1] = image_g[idx1];
    rgba[2] = image_b[idx1];
    rgba[3] = image_a[idx1];

    float v_pos[3];
    v_pos[0] = av_pos[idx1 + 0];
    v_pos[1] = av_pos[idx1 + 1];
    v_pos[2] = av_pos[idx1 + 2];

    /* We integrate our ray */

    int cur_ind[3], step[3], x, y, i, direction;
    float intersect_t = 1.0;
    float intersect[3], tmax[3], tdelta[3];
    float enter_t, tr, tl, temp_x, temp_y;

    int offset;
    int i0 = 0;

    for (i = 0; i < 3; i++)
    {
        step[i] = ((v_dir[i] < 0) ? -1 : 1);
        x = (i + 1) % 3;
        y = (i + 2) % 3;
        tl = (tg.left_edge[i] - v_pos[i])/v_dir[i];
        tr = (tg.right_edge[i] - v_pos[i])/v_dir[i];
        temp_x = (v_pos[i] + tl*v_dir[x]);
        temp_y = (v_pos[i] + tl*v_dir[y]);

        if( (tg.left_edge[x] <= temp_x) &&
            (temp_x <= tg.right_edge[x]) &&
            (tg.left_edge[y] <= temp_y) &&
            (temp_y <= tg.right_edge[y]) &&
            (0.0 <= tl) && (tl < intersect_t) ) intersect_t = tl;

        temp_x = (v_pos[x] + tr*v_dir[x]);
        temp_y = (v_pos[y] + tr*v_dir[y]);

        if( (tg.left_edge[x] <= temp_x) &&
            (temp_x <= tg.right_edge[x]) &&
            (tg.left_edge[y] <= temp_y) &&
            (temp_y <= tg.right_edge[y]) &&
            (0.0 <= tr) && (tr < intersect_t) ) intersect_t = tr;

    }

    for (i = 0; i < 3; i++)
    {
        if ( (tg.left_edge[i] <= v_pos[i]) &&
             (v_pos[i] <= tg.right_edge[i])) i0++;
    }
    if (i0 == 3) intersect_t = 0.0;

    if((intersect_t < 0) || (intersect_t > 1.0)) return;

    for (i = 0; i < 3;  i++)
    {
        intersect[i] = v_pos[i] + intersect_t * v_dir[i];
        cur_ind[i] = (int) floor((intersect[i] +
                                  step[i]*1e-7*tg.dds[i] -
                                  tg.left_edge[i])/tg.dds[i]);
        tmax[i] = (((cur_ind[i]+step[i])*tg.dds[i])+
                     tg.left_edge[i]-v_pos[i])/v_dir[i];
        if((cur_ind[i] == tg.dims[i]) && (step[i] < 0)) cur_ind[i] -= 1;
        if((cur_ind[i] < 0) || (cur_ind[i] >= tg.dims[i])) return;
        if(step[i] > 0) offset = 1;
        if(step[i] < 0) offset = 0;
        tmax[i] = (((cur_ind[i]+offset)*tg.dds[i])+tg.left_edge[i]-v_pos[i])/v_dir[i];
        tdelta[i] = abs(tg.dds[i]/v_dir[i]);
    }
    enter_t = intersect_t;

    /* This is the primary grid walking loop */
    while (1) {
        if(((cur_ind[0] < 0) || (cur_ind[0] >= tg.dims[0]))
          ||(cur_ind[1] < 0) || (cur_ind[1] >= tg.dims[1])
          ||(cur_ind[2] < 0) || (cur_ind[2] >= tg.dims[2])) break;
        if (tmax[0] < tmax[1]) {
            if (tmax[0] < tmax[2]) {
                direction = 0;
            } else {
                direction = 2;
            }
        } else {
            if (tmax[1] < tmax[2]) {
                direction = 1;
            } else {
                direction = 2;
            }
        }
        sample_values(v_pos, v_dir, enter_t, tmax[direction],
                      cur_ind, rgba, tf, tg);
        cur_ind[direction] += step[direction];
        enter_t = tmax[direction];
        tmax[direction] += tdelta[direction];
    }

}
